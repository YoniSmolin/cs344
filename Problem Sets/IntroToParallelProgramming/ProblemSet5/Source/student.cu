#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"

__global__
void Serial_SharedMem(const unsigned int* const vals, unsigned int* const histo, const unsigned int numBins, const unsigned int numElems)
{	
	extern __shared__ unsigned int histogram[];	

	for (unsigned int i = 0; i < numBins; i++) histogram[i] = 0;

	for (unsigned int i = 0; i < numElems; i++) histogram[vals[i]]++;

	for (unsigned int i = 0; i < numBins; i++) histo[i] = histogram[i];
}

void computeHistogram(const unsigned int* const d_vals, unsigned int* const d_histo, const unsigned int numBins, const unsigned int numElems)
{
	Serial_SharedMem << <1, 1, numBins * sizeof(unsigned int) >> > (d_vals, d_histo, numBins, numElems);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
